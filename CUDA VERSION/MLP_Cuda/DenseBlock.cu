#include "hip/hip_runtime.h"
#include "DenseBlock.cuh"


// ======== DENSE LAYER ======== //
DenseBlock::DenseBlock(const int& n_inputs, const int& n_neurons) : m_weights(n_inputs + 1, n_neurons) {

	// Xavier initialization
	double limit = std::sqrt(6.0 / (n_inputs + n_neurons));

	for (size_t i = 0; i < n_neurons; i++) {
		for (size_t j = 0; j < n_inputs; j++) {
			m_weights[j][i] = random(-limit, limit);
		}
		m_weights[n_inputs][i] = random(-limit, limit);
	}
};

void DenseBlock::activate(const Matrix& inputs) {
	for (int i = 0; i < inputs.size(); i++)
		for (int j = 0; j < inputs[0].size(); j++)
			m_output[i][j] = std::max(0.0, inputs[i][j]);
}


void DenseBlock::softmax_activation(const Matrix& inputs) {

	dvector maxs(inputs.size());
	for (int i = 0; i < inputs.size(); i++) {
		maxs[i] = inputs[i][0];
		for (int j = 0; j < inputs[0].size(); j++)
			if (inputs[i][j] > maxs[i])
				maxs[i] = inputs[i][j];
	}

	Matrix expvalues = inputs;
	dvector sum_of_exps(inputs.size(), 0);
	for (int i = 0; i < inputs.size(); i++) {
		for (int j = 0; j < inputs[0].size(); j++) {
			expvalues[i][j] = pow(EULERS_NUMBER, inputs[i][j] - maxs[i]);
			sum_of_exps[i] += expvalues[i][j];
		}
	}

	for (int i = 0; i < inputs.size(); i++)
		for (int j = 0; j < inputs[0].size(); j++)
			m_output[i][j] = expvalues[i][j] / sum_of_exps[i]; // m_output = probabilities

}


void DenseBlock::forward(Matrix inputs, const std::string& softmax) {
	m_preactivation = inputs.addBias() * m_weights;

	m_output = m_preactivation;
	(softmax == "softmax" ? softmax_activation(m_output) : activate(m_output));
};

Matrix DenseBlock::output() {
	return m_output;
}
Matrix DenseBlock::weights() {
	return m_weights;
}
void DenseBlock::setWeights(const Matrix& weights) {
	m_weights = weights;
}
Matrix DenseBlock::preactivation() {
	return m_preactivation;
}