#include "hip/hip_runtime.h"
#include "MLP.cuh"

MLP::MLP(const hyperparameters& hyper) : _hyperparameters(hyper), L(_hyperparameters.hidden_layers_sizes.size() + 1) {

    std::vector<Matrix> matrix_L_sized(L);
    m_deltas = matrix_L_sized;
    m_grad_J = matrix_L_sized;

    dvector hidden_layers_sizes = _hyperparameters.hidden_layers_sizes;
    hidden_layers_sizes.insert(hidden_layers_sizes.begin(), _hyperparameters.input_dim);
    hidden_layers_sizes.push_back(_hyperparameters.output_dim);
    for (int l = 0; l < L; l++) {
        DenseBlock layer(hidden_layers_sizes[l], hidden_layers_sizes[l + 1]);
        m_layers.push_back(layer);

        M.push_back(Matrix(hidden_layers_sizes[l] + 1, hidden_layers_sizes[l + 1]));
        V.push_back(M[l]);
    }
    t = 0;

};

Matrix MLP::forward(Matrix input) {

    m_layers[0].forward(input);
    for (int l = 1; l < L - 1; l++)
        m_layers[l].forward(_hyperparameters.learn ? m_layers[l - 1].output().dropoutMask(_hyperparameters.dropout_rate) : m_layers[l - 1].output());
    m_layers[L - 1].forward(m_layers[L - 2].output(), "softmax");

    return m_layers.back().output();

}

void MLP::backpropagation(Matrix input, Matrix y_hot_one) {

    m_deltas[L - 1] = m_layers.back().output() - y_hot_one;
    m_grad_J[L - 1] = m_layers[L - 2].output().addBias_then_T() * m_deltas[L - 1];

    for (int l = L - 2; l >= 0; l--) {
        m_deltas[l] = (m_deltas[l + 1] * m_layers[l + 1].weights().T_then_removeBias()).hadamard(m_layers[l].preactivation().derivReLU());
        m_grad_J[l] = (l == 0 ? input : m_layers[l - 1].output()).addBias_then_T() * m_deltas[l];
    }

}

// ========================================================== ADAM OPTIMIZER IMPLEMENTATION
__global__ void adamKernel(double* W, double* M, double* V, const double* G, int N, hyperparameters hyp, int t) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    M[idx] = hyp.adam_beta_m * M[idx] + (1.0 - hyp.adam_beta_m) * G[idx];
    V[idx] = hyp.adam_beta_v * V[idx] + (1.0 - hyp.adam_beta_v) * G[idx] * G[idx];

    double m_hat = M[idx] / (1 - pow(hyp.adam_beta_m, t));
    double v_hat = V[idx] / (1 - pow(hyp.adam_beta_v, t));

    W[idx] -= hyp.learning_rate * (m_hat / (sqrt(v_hat) + 1e-8));
}
void MLP::Adam() {
    t += 1;

    // Implementation of CUDA framework for backprop
    for (int l = 0; l < L; ++l) {
        Matrix& W_mat = m_layers[l].weights();
        Matrix& M_mat = M[l];
        Matrix& V_mat = V[l];
        Matrix& G_mat = m_grad_J[l];

        int rows = static_cast<int>(W_mat.size());
        int cols = static_cast<int>(W_mat[0].size());
        int N = rows * cols;
        size_t bytes = N * sizeof(double);

        std::vector<double> flat_W; flat_W.reserve(N);
        std::vector<double> flat_M; flat_M.reserve(N);
        std::vector<double> flat_V; flat_V.reserve(N);
        std::vector<double> flat_G; flat_G.reserve(N);

        for (auto& row : W_mat) flat_W.insert(flat_W.end(), row.begin(), row.end());
        for (auto& row : M_mat) flat_M.insert(flat_M.end(), row.begin(), row.end());
        for (auto& row : V_mat) flat_V.insert(flat_V.end(), row.begin(), row.end());
        for (auto& row : G_mat) flat_G.insert(flat_G.end(), row.begin(), row.end());

        double* d_W, * d_M, * d_V, * d_G;
        hipMalloc(&d_W, bytes); hipMalloc(&d_M, bytes);
        hipMalloc(&d_V, bytes); hipMalloc(&d_G, bytes);

        hipMemcpy(d_W, flat_W.data(), bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_M, flat_M.data(), bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_V, flat_V.data(), bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_G, flat_G.data(), bytes, hipMemcpyHostToDevice);

        int TPB = 256;
        int nBlocks = (N + TPB - 1) / TPB;
        adamKernel<<<nBlocks, TPB>>>(d_W, d_M, d_V, d_G, N, _hyperparameters, t);
        hipDeviceSynchronize();

        std::vector<double> new_W(N), new_M(N), new_V(N);
        hipMemcpy(new_W.data(), d_W, bytes, hipMemcpyDeviceToHost);
        hipMemcpy(new_M.data(), d_M, bytes, hipMemcpyDeviceToHost);
        hipMemcpy(new_V.data(), d_V, bytes, hipMemcpyDeviceToHost);

        for (int i = 0; i < rows; ++i) {
            for (int j = 0; j < cols; ++j) {
                int idx = i * cols + j;
                W_mat[i][j] = new_W[idx];
                M_mat[i][j] = new_M[idx];
                V_mat[i][j] = new_V[idx];
            }
        }
        m_layers[l].setWeights(W_mat);

        // Free memory
        hipFree(d_W);
        hipFree(d_M);
        hipFree(d_V);
        hipFree(d_G);
    }
}


double MLP::CELoss(Matrix y_hot_one) {
    double loss = CELossFunction(m_layers.back().output(), y_hot_one).second;
    return loss;
}

double MLP::backwards(Matrix input, Matrix y_hot_one) {
    if (t == 0) {
        print("Epochs: Training Loss");
        print(t, ": ", CELoss(y_hot_one));
    }

    backpropagation(input, y_hot_one);
    Adam();

    double loss = CELoss(y_hot_one);
    if (t % 50 == 0)
        print(t, ": ", loss);
    return loss;
}



void MLP::saveWeights(const std::string& filename) {
    std::ofstream file(filename);
    for (auto& layer : m_layers) {
        Matrix W = layer.weights();
        for (auto& row : W) {
            for (double val : row)
                file << val << " ";
            file << "\n";
        }
        file << "===\n"; // S�parateur entre couches
    }
    file.close();
}

void MLP::loadWeights(const std::string& filename) {
    std::ifstream file(filename);
    std::string line;
    int layer_index = 0;
    Matrix W;
    while (std::getline(file, line)) {
        if (line == "===") {
            m_layers[layer_index].setWeights(W);
            W.clear();
            layer_index++;
        }
        else {
            std::istringstream iss(line);
            dvector row;
            double val;
            while (iss >> val)
                row.push_back(val);
            W.push_back(row);
        }
    }
    file.close();
}
