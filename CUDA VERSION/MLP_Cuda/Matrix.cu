#include "hip/hip_runtime.h"
﻿#include "Matrix.cuh"
#include <hiprand/hiprand_kernel.h>

Matrix::Matrix() {}
Matrix::Matrix(size_t row, size_t columns) : _rows(row), _cols(columns), dmatrix(row, dvector(columns, 0.0)) {}
Matrix::Matrix(std::initializer_list<std::initializer_list<double>> init) {
	*this = init;
}
Matrix::Matrix(dmatrix init) {

	this->clear();
	for (auto& row : init)
		this->emplace_back(row);

	_rows = this->size();
	_cols = (*this)[0].size();
}
void Matrix::operator=(std::initializer_list<std::initializer_list<double>> init) {

	this->clear();
	for (auto& row : init)
		this->emplace_back(row);

	_rows = this->size();
	_cols = (*this)[0].size();
}

// ================================================================================================================== MULTIPLICATION
__global__ void multKernel(double* A, double* B, double* C, int A_rows, int A_cols, int B_cols) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < A_rows && col < B_cols) {
		double sum = 0.0;
		for (int k = 0; k < A_cols; k++) {
			sum += A[row * A_cols + k] * B[k * B_cols + col];
		}
		C[row * B_cols + col] = sum;
	}
}
Matrix Matrix::operator*(const Matrix& B) {
	size_t B_cols = B[0].size();
	Matrix C(_rows, B_cols);

	// Device allocs
	double* d_A, * d_B, * d_C;
	size_t size_A = _rows * _cols * sizeof(double);
	size_t size_B = _cols * B_cols * sizeof(double);
	size_t size_C = _rows * B_cols * sizeof(double);
	hipMalloc(&d_A, size_A);
	hipMalloc(&d_B, size_B);
	hipMalloc(&d_C, size_C);

	// Flatten matrices (switch to vector later on)
	std::vector<double> flat_A, flat_B;
	for (auto& row : *this) flat_A.insert(flat_A.end(), row.begin(), row.end());
	for (auto& row : B) flat_B.insert(flat_B.end(), row.begin(), row.end());

	// Memory copy from host to device
	hipMemcpy(d_A, flat_A.data(), size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_B, flat_B.data(), size_B, hipMemcpyHostToDevice);

	// Spit dimensions
	dim3 threadsPerBlock(16, 16); dim3 blocksPerGrid((B_cols + 15) / 16, (_rows + 15) / 16);
	multKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, _rows, _cols, B_cols); // Make operation

	// Paste operation result to the host
	std::vector<double> flat_C(_rows * B_cols);
	hipMemcpy(flat_C.data(), d_C, size_C, hipMemcpyDeviceToHost);
	for (size_t i = 0; i < _rows; ++i)
		for (size_t j = 0; j < B_cols; ++j)
			C[i][j] = flat_C[i * B_cols + j];

	hipFree(d_A); hipFree(d_B); hipFree(d_C); // Free allocs
	return C;
}


// ================================= HADAMARD
__global__ void hadamardKernel(const double* A, const double* B, double* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) C[idx] = A[idx] * B[idx];
}
Matrix Matrix::hadamard(const Matrix& B) {
	return runGPUBinary("hadamardKernel", B);
}

// ================================= SCALAR MULTIPLICATION
__global__ void scalarMulKernel(const double* A, double* C, double a, int size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) C[idx] = A[idx] * a;
}
Matrix Matrix::operator*(const double& a) {
	return runGPUUnary("scalar multiplication", a);
}


// ================================= ADDITION
__global__ void addKernel(const double* A, const double* B, double* C, int size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) C[idx] = A[idx] + B[idx];
}
Matrix Matrix::operator+(const Matrix& B) {
	return runGPUBinary("addKernel", B);
}

// ================================= SUBSTRACTION
__global__ void subKernel(const double* A, const double* B, double* C, int size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) C[idx] = A[idx] - B[idx];
}
Matrix Matrix::operator-(const Matrix& B) {
	return runGPUBinary("subKernel", B);
}

// ================================= TRANSPOSITION
__global__ void transposeKernel(const double* A, double* C, int rows, int cols) {
	int i = blockDim.y * blockIdx.y + threadIdx.y; // row
	int j = blockIdx.x * blockDim.x + threadIdx.x; // col

	if (i < rows && j < cols)
		C[j * rows + i] = A[i * cols + j];
}
Matrix Matrix::T() {
	return runGPUUnary("transposition");
}

// ================================= ADD BIASES
__global__ void addBiasKernel(const double* A, double* C, int rows, int cols) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < rows) {
		for (int j = 0; j < cols; ++j)
			C[i * (cols + 1) + j] = A[i * cols + j];

		C[i * (cols + 1) + cols] = 1.0;
	}
}
Matrix Matrix::addBias() {
	return runGPUUnary("add bias");
}

// ================================= ADD BIASES AND THEN TRANSPOSE
__global__ void addBiasAndTKernel(const double* A, double* C, int rows, int cols) {
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < cols && j < rows)
		C[i * rows + j] = A[j * cols + i];
	if (i == cols && j < rows)
		C[cols * rows + j] = 1.0;
}
Matrix Matrix::addBias_then_T() {
	return runGPUUnary("add bias then transpose");
}

// ================================= TRANSPOSE THEN REMOVE BIASES
__global__ void TAndRemoveBiasKernel(const double* A, double* C, int rows, int cols) {
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < cols && j < rows - 1)
		C[i * (rows - 1) + j] = A[j * cols + i];
}
Matrix Matrix::T_then_removeBias() {
	return runGPUUnary("transpose and remove bias");
}



// ================================= DROPOUT
__global__ void dropoutKernel(const double* A, double* C, double keep_prob, int size, unsigned long seed) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size) return;

	hiprandState state;
	hiprand_init(seed, idx, 0, &state);

	double randVal = hiprand_uniform_double(&state);
	C[idx] = (randVal > keep_prob) ? 0.0 : (A[idx] / keep_prob);
}
Matrix Matrix::dropoutMask(const double& dropout) {
	return runGPUUnary("dropout", dropout);
}

// ================================= APPLY ReLU DERIVATE
__global__ void derivReLUKernel(const double* A, double* C, int size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		C[idx] = (A[idx] > 0.0) ? 1.0 : 0.0;
}
Matrix Matrix::derivReLU() {
	return runGPUUnary("derivate of ReLU");
}


// Set Maximum of vectors to 1 and others to 0.
__global__ void getMaxKernel(const double* A, double* C, int rows, int cols) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < rows) {
		double maxVal = A[i * cols];
		int maxIdx = 0;

		for (int j = 1; j < cols; ++j) {
			double val = A[i * cols + j];
			if (val > maxVal) {
				maxVal = val;
				maxIdx = j;
			}
		}

		for (int j = 0; j < cols; ++j)
			C[i * cols + j] = (j == maxIdx) ? 1.0 : 0.0;
	}

}

Matrix Matrix::setMaxToOne() {
	return runGPUUnary("set max to one and other to zero");
}


Matrix Matrix::runGPUBinary(const std::string& operation, const Matrix& B) {
	int N = _rows * _cols;
	Matrix C(_rows, _cols);

	// device allocations
	double* d_A, * d_B, * d_C;
	size_t bytes = N * sizeof(double);
	hipMalloc(&d_A, bytes);
	hipMalloc(&d_B, bytes);
	hipMalloc(&d_C, bytes);

	// flat copy (may implement Matrices as Vectors later on
	std::vector<double> flat_A; flat_A.reserve(N);
	for (auto& row : *this)
		flat_A.insert(flat_A.end(), row.begin(), row.end());
	std::vector<double> flat_B; flat_B.reserve(N);
	for (auto& row : B)
		flat_B.insert(flat_B.end(), row.begin(), row.end());

	// Memory Copy from Host to Device
	hipMemcpy(d_A, flat_A.data(), bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, flat_B.data(), bytes, hipMemcpyHostToDevice);

	// Kernel launch in 1D
	int TPB = 256;
	int nBlocks = (N + TPB - 1) / TPB;
	if (operation == "hadamardKernel")
		hadamardKernel<<<nBlocks, TPB>>>(d_A, d_B, d_C, N);
	if (operation == "addKernel")
		addKernel<<<nBlocks, TPB>>>(d_A, d_B, d_C, N);
	if (operation == "subKernel")
		subKernel<<<nBlocks, TPB>>>(d_A, d_B, d_C, N);

	// Get the flat vector out of the kernel
	std::vector<double> flat_C(N);
	hipMemcpy(flat_C.data(), d_C, bytes, hipMemcpyDeviceToHost);
	for (size_t i = 0; i < _rows; ++i)
		for (size_t j = 0; j < _cols; ++j)
			C[i][j] = flat_C[i * _cols + j];

	// Free allocations
	hipFree(d_A); hipFree(d_B); hipFree(d_C); return C;
}

Matrix Matrix::runGPUUnary(const std::string& operation, const double& a) {
	int C_rows = _rows;
	int C_cols = _cols;
	if (operation == "add bias")
		C_cols = _cols + 1;
	if (operation == "transposition") {
		C_rows = _cols;
		C_cols = _rows;
	}
	if (operation == "add bias then transpose") {
		C_rows = _cols + 1;
		C_cols = _rows;
	}
	if (operation == "transpose and remove bias") {
		C_rows = _cols;
		C_cols = _rows - 1;
	}

	int N = C_rows * C_cols;
	Matrix C(C_rows, C_cols);

	double* d_A, * d_C;
	size_t size_A = _rows * _cols * sizeof(double);
	size_t size_C = N * sizeof(double);

	hipMalloc(&d_A, size_A);
	hipMalloc(&d_C, size_C);

	std::vector<double> flat_A; flat_A.reserve(N);
	for (auto& row : *this) flat_A.insert(flat_A.end(), row.begin(), row.end());

	hipMemcpy(d_A, flat_A.data(), size_A, hipMemcpyHostToDevice);

	// Thread and block sizes for 2D and 1D Kernel.
	int TPB = 256;
	int nBlocks = (N + TPB - 1) / TPB;

	dim3 blockSize(16, 16);
	dim3 gridSize((C_cols + blockSize.x - 1) / blockSize.x,
				  (C_rows + blockSize.y - 1) / blockSize.y);
	if(operation == "scalar multiplication")
		scalarMulKernel<<<nBlocks, TPB>>>(d_A, d_C, a, _rows * _cols);
	if(operation == "transposition")
		transposeKernel<<<gridSize, blockSize>>>(d_A, d_C, _rows, _cols);
	if(operation == "add bias")
		addBiasKernel<<<nBlocks, TPB>>>(d_A, d_C, _rows, _cols);
	if(operation == "add bias then transpose")
		addBiasAndTKernel<<<gridSize, blockSize>>>(d_A, d_C, _rows, _cols);
	if(operation == "transpose and remove bias")
		TAndRemoveBiasKernel<<<gridSize, blockSize>>>(d_A, d_C, _rows, _cols);
	if (operation == "dropout")
		dropoutKernel<<<nBlocks, TPB>>>(d_A, d_C, 1.0 - a, _rows * _cols, time(NULL));
	if (operation == "derivate of ReLU")
		derivReLUKernel<<<nBlocks, TPB>>>(d_A, d_C, _rows * _cols);
	if (operation == "set max to one and other to zero")
		getMaxKernel<<<nBlocks, TPB>>>(d_A, d_C, _rows, _cols);

	// Copy result back
	std::vector<double> flat_C(C_rows * C_cols);
	hipMemcpy(flat_C.data(), d_C, size_C, hipMemcpyDeviceToHost);

	// Fill C matrix
	for (size_t i = 0; i < C_rows; ++i)
		for (size_t j = 0; j < C_cols; ++j)
			C[i][j] = flat_C[i * C_cols + j];

	// Cleanup
	hipFree(d_A); hipFree(d_C);

	return C;
}

Matrix Matrix::runGPUUnary(const std::string& operation) {
	return runGPUUnary(operation, 0.0);
}