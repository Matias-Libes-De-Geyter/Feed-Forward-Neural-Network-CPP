#include "hip/hip_runtime.h"
#include "functions.cuh"

double random(const double& min, const double& max) {
	std::mt19937_64 rng{};
	rng.seed(std::random_device{}());
	return std::uniform_real_distribution<>{min, max}(rng);
}
Matrix hotOne(const dvector& y, const int& nElements) {

	Matrix C(y.size(), nElements);

	for (int i = 0; i < y.size(); i++)
		C[i][y[i]] = 1;

	return C;
}


// ====================================================================== CROSS ENTROPY LOSS GPU IMPLEMENTATION
__global__ void cross_entropy_kernel(const double* y_pred, const double* y_true, double* loss, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < rows)
        for (int j = 0; j < cols; ++j)
            if (y_true[i * cols + j] == 1.0) {
                double prob = fmax(y_pred[i * cols + j], 1e-9);
                loss[i] = -log(prob);
                break;
            }

}

std::pair<dvector, double> CELossFunction(const Matrix& y_pred, const Matrix& y_true) {
    int rows = static_cast<int>(y_pred.size());
    int cols = static_cast<int>(y_pred[0].size());

    dvector y_pred_flat(rows * cols);
    dvector y_true_flat(rows * cols);

    for (int i = 0; i < rows; ++i)
        for (int j = 0; j < cols; ++j) {
            y_pred_flat[i * cols + j] = y_pred[i][j];
            y_true_flat[i * cols + j] = y_true[i][j];
        }

    double *d_y_pred, *d_y_true, *d_loss;
    hipMalloc(&d_y_pred, sizeof(double) * rows * cols);
    hipMalloc(&d_y_true, sizeof(double) * rows * cols);
    hipMalloc(&d_loss, sizeof(double) * rows);

    hipMemcpy(d_y_pred, y_pred_flat.data(), sizeof(double) * rows * cols, hipMemcpyHostToDevice);
    hipMemcpy(d_y_true, y_true_flat.data(), sizeof(double) * rows * cols, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (rows + blockSize - 1) / blockSize;
    cross_entropy_kernel<<<gridSize, blockSize>>>(d_y_pred, d_y_true, d_loss, rows, cols);
    hipDeviceSynchronize();

    dvector loss(rows, 1e-9);
    hipMemcpy(loss.data(), d_loss, sizeof(double) * rows, hipMemcpyDeviceToHost);

    double mean_loss = 0.0;
    for (double val : loss) mean_loss += val;
    mean_loss /= rows;

    // Cleanup
    hipFree(d_y_pred);
    hipFree(d_y_true);
    hipFree(d_loss);

    return { loss, mean_loss };
}