﻿#include "TrainerClassifier.cuh"
#pragma GCC diagnostic ignored "-Wnarrowing"


hyperparameters current_hyperparameters = {
    28 * 28,                  // input_dim
    10,                       // output_dim
    {256, 128},               // hidden_layers_sizes
    0,                        // epochs (0 = all the database)
    32,                       // mini_batch_size
    0.001,                    // learning_rate
    0.9,                      // adam_beta_m
    0.999,                    // adam_beta_v
    0.001,                    // dropout_rate
    false,                    // early_stopping
    150,                      // patience
    false,                    // learn
    true,                     // test
    false                      // store_data
};

int main() {
    // Ask for model training
    print("Train ? (y/n)"); char a; std::cin >> a;
    if (a == 'y') current_hyperparameters.learn = true;

    // MLP init
    MLP NN(current_hyperparameters);

    // Training
    if (current_hyperparameters.learn) {
        train(NN, current_hyperparameters);
        NN.saveWeights("model_weights.txt");
    }
    else {
        NN.loadWeights("model_weights.txt");
        print("Weights loaded !");
    }

    if (current_hyperparameters.test) test(NN, current_hyperparameters);

    return 0;
}